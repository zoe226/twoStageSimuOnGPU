#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <cmath>
#include <hipfft/hipfft.h>
#define M_PI 3.141592653589793

__global__ void rectWin_kernel(float* winCoef, uint16_t winLen)
{
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    if(tidx < winLen)
    {
        winCoef[tidx] = 1.0;
    }
}

__global__ void hanningWin_kernel(float* winCoef, uint16_t winLen)
{
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    if(tidx < winLen){
        winCoef[tidx] = 0.5*(1 - cos(2 * M_PI * tidx / (winLen - 1)));
    }
}

__global__ void hammingWin_kernel(float* winCoef, uint16_t winLen)
{
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    if (tidx < winLen)
    {
        winCoef[tidx] = 0.54 - 0.46 * cos(2 * M_PI * tidx / (winLen - 1));
    }
}

__global__ void addWin_kernel(float* WinRout_RangeSampleNum_ChirpNum_RxNum_device, int16_t* radarInputdata_device, float* win_Coef_device, uint16_t RangeSampleNum, uint16_t ChirpNum, uint16_t RxNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tix = threadIdx.z + blockDim.z * blockIdx.z;

    if (tix < RangeSampleNum && tiy < RxNum && tiz < ChirpNum)
    {
        WinRout_RangeSampleNum_ChirpNum_RxNum_device[tix + tiz * RangeSampleNum + tiy * RangeSampleNum * ChirpNum] = radarInputdata_device[tix + tiy * RangeSampleNum + tiz * RangeSampleNum * RxNum] * win_Coef_device[tix];
    } 
}

__global__ void dataReshape_afterFFTR_kernel(float2* CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum, float2* CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device, uint16_t* all_tx_seq_pos_device, uint16_t TxNum, uint16_t RxNum, uint16_t TxReuseNum, uint16_t CoarseRangeNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;
    int velocityNum = TxNum * TxReuseNum;
    if(tix < CoarseRangeNum && tiy < velocityNum && tiz < RxNum)
    {
        int txIdx = tiy / TxReuseNum;
        int chirpLoc = tiy % TxReuseNum;
        int chirpIdx = all_tx_seq_pos_device[chirpLoc + txIdx * TxReuseNum] - 1;
        CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum[chirpIdx + tix * velocityNum + tiz * velocityNum * CoarseRangeNum] = CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device[tix + chirpIdx * (CoarseRangeNum + 1) + tiz * (CoarseRangeNum + 1) * VelocityNum];
    }
}

__global__ void addWinD_kernel(float2* WinDout_ChirpNum_RangeSampleNum_MIMONum_device, float2* CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device, float* win_Coef_device, uint16_t VelocityNum, uint16_t CoarseRangeNum, uint16_t MimoNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;

    if (tix < VelocityNum && tiy < CoarseRangeNum && tiz < MimoNum)
    {
        WinDout_ChirpNum_RangeSampleNum_MIMONum_device[tix + tiy * VelocityNum + tiz * VelocityNum * CoarseRangeNum].x = CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device[tix + tiy * VelocityNum + tiz * VelocityNum * CoarseRangeNum].x * win_Coef_device[tix];
        WinDout_ChirpNum_RangeSampleNum_MIMONum_device[tix + tiy * VelocityNum + tiz * VelocityNum * CoarseRangeNum].y = CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device[tix + tiy * VelocityNum + tiz * VelocityNum * CoarseRangeNum].y * win_Coef_device[tix];
    } 
}

__global__ void fftshift(float2* indata, uint16_t dim1, uint16_t dim2, uint16_t dim3)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;
    float2 temp;
    if (tix < dim1/2 && tiy < dim2 && tiz < dim3)
    {
        temp.x = indata[tix + tiy * dim1 + tiz * dim1 * dim2].x;
        temp.y = indata[tix + tiy * dim1 + tiz * dim1 * dim2].y;

        indata[tix + tiy * dim1 + tiz * dim1 * dim2].x = indata[tix + dim1 / 2 + tiy * dim1 + tiz * dim1 * dim2].x;
        indata[tix + tiy * dim1 + tiz * dim1 * dim2].y = indata[tix + dim1 / 2 + tiy * dim1 + tiz * dim1 * dim2].y;

        indata[tix + dim1 / 2 + tiy * dim1 + tiz * dim1 * dim2].x = temp.x;
        indata[tix + dim1 / 2 + tiy * dim1 + tiz * dim1 * dim2].y = temp.y;
    }
}

__global__ void spatial_reorder_kernel(float2* result_xNum_yNum_VeloFFTNum_RangeNum, float2* FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device, uint16_t* pos_in_mat, uint16_t MimoNum, uint16_t VelocityNum, uint16_t CoarseRangeNum, uint16_t VirtArrHorGridLen, uint16_t VirtArrVertGridLen)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;

    if (tix < VelocityNum && tiy < CoarseRangeNum && tiz < MimoNum)
    {
        int xIdx = pos_in_mat[2 * tiz];
        int yIdx = pos_in_mat[2 * tiz + 1];
        result_xNum_yNum_VeloFFTNum_RangeNum[xIdx + yIdx * VirtArrHorGridLen + tix * VirtArrHorGridLen * VirtArrVertGridLen + tiy * VirtArrHorGridLen * VirtArrVertGridLen * VelocityNum] = FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device[tix + tiy * VelocityNum + tiz * VelocityNum * CoarseRangeNum];
    }
}

__global__ void addWinA_kernel(float2* winAout_xNum_yNum_VeloFFTNum_RangeNum_device, float2* result_xNum_yNum_VeloFFTNum_RangeNum_device, float* win_Coef_device, uint16_t VirtArrHorGridLen, uint16_t VirtArrVertGridLen, uint16_t VelocityNum, uint16_t CoarseRangeNum, uint16_t AngleHorNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;
    if (tix < VirtArrHorGridLen && tiy < VelocityNum && tiz < CoarseRangeNum)
    {
        winAout_xNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = result_xNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * VirtArrHorGridLen * VirtArrVertGridLen + tiz * VirtArrHorGridLen * VirtArrVertGridLen * VelocityNum].x * win_Coef_device[tix];
        winAout_xNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = result_xNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * VirtArrHorGridLen * VirtArrVertGridLen + tiz * VirtArrHorGridLen * VirtArrVertGridLen * VelocityNum].y * win_Coef_device[tix];
        winAout_xNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum * (VirtArrVertGridLen - 1)  + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = result_xNum_yNum_VeloFFTNum_RangeNum_device[tix + VirtArrHorGridLen * (VirtArrVertGridLen - 1) + tiy * VirtArrHorGridLen * VirtArrVertGridLen + tiz * VirtArrHorGridLen * VirtArrVertGridLen * VelocityNum].x * win_Coef_device[tix];
        winAout_xNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum * (VirtArrVertGridLen - 1)  + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = result_xNum_yNum_VeloFFTNum_RangeNum_device[tix + VirtArrHorGridLen * (VirtArrVertGridLen - 1) + tiy * VirtArrHorGridLen * VirtArrVertGridLen + tiz * VirtArrHorGridLen * VirtArrVertGridLen * VelocityNum].y * win_Coef_device[tix];
    }
}

__global__ void fftshift_ffta(float2* SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, uint16_t AngleHorNum, uint16_t VirtArrVertGridLen, uint16_t VelocityNum, uint16_t CoarseRangeNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;
    int tiz = threadIdx.z + blockDim.z * blockIdx.z;

    float2 temp1;
    float2 temp2;
    if(tix < AngleHorNum/2 && tiy < VelocityNum && tiz < CoarseRangeNum)
    {
        temp1.x = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x;
        temp1.y = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y;
        
        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x;
        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y;        

        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = temp1.x;
        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = temp1.y;

        temp2.x = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x;
        temp2.y = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y;

        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x;
        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y;

        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].x = temp2.x;
        SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tix + AngleHorNum / 2 + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum].y = temp2.y;
    }
}

__global__ void ffta_abs_kernel(float* SpatialFFTA_ABS_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, float2* SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, uint16_t AngleHorNum, uint16_t VirtArrVertGridLen, uint16_t VelocityNum, uint16_t CoarseRangeNum)
{
    int tix = threadIdx.x + blockDim.x * blockIdx.x;
    int tiy = threadIdx.y + blockDim.y * blockIdx.y;

    if (tix < AngleHorNum && tiy < VelocityNum && tiz < CoarseRangeNum)
    {
        int index1 = tix + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum;
        int index2 = tix + (VirtArrVertGridLen - 1) * AngleHorNum + tiy * AngleHorNum * VirtArrVertGridLen + tiz * AngleHorNum * VirtArrVertGridLen * VelocityNum;
        int index3 = tix + tiy * AngleHorNum + tiz * AngleHorNum * VelocityNum;
        float y1 = sqrt(powf(SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[index1].x,2) + powf(SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[index1].y,2));
        float y2 = sqrt(powf(SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[index2].x,2) + powf(SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[index2].y,2));
        SpatialFFTA_ABS_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[index3] = (y1 + y2) / VirtArrVertGridLen;
    }
}

__global__ void cfar3d_cal_across_ArbitaryDim_kernel(float* SpatialFFTVelSel_VeloNum_RangeNum_device, float* SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, uint16_t AngleHorNum, uint16_t VelocityNum, uint16_t CoarseRangeNum)
{
    int tiy = blockDim.y * blockIdx.y + threadIdx.y;
    int tiz = blockDim.z * blockIdx.z + threadIdx.z;
    if (tiy < VelocityNum && tiz < CoarseRangeNum)
    {
        float temp = SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tiy * AngleHorNum + tiz * VelocityNum * AngleHorNum];
        for (size_t i = 0; i < AngleHorNum; i++)
        {
            if (temp < SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[i + tiy * AngleHorNum + tiz * VelocityNum * AngleHorNum])
            {
                temp = SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[i + tiy * AngleHorNum + tiz * VelocityNum * AngleHorNum];
            }
        }
        SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device[tiy + tiz * VelocityNum] = temp;
    }
}

__global__ void peak_search_kernel(uint8_t* isPeak, float* SpatialFFTVelSel_VeloNum_RangeNum_device, uint16_t DetectCell_RIndex_Min_u10, uint16_t DetectCell_RIndex_Max_u10, uint16_t DetectCell_VIndex_Min_u11, uint16_t DetectCell_VIndex_Max_u11, uint16_t ChirpNum_u11, uint16_t RangeCellNum_u10)
{
    // peak search enable and peakSearchWin = 1
    int tix = blockDim.x * blockIdx.x + threadIdx.x;
    int tiy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tix < DetectCell_RIndex_Max_u10 && tix >= (DetectCell_RIndex_Min_u10 - 1) && tiy < DetectCell_VIndex_Max_u11 && tiy >= (DetectCell_VIndex_Min_u11-1))
    {
        int V_Upboundary_u11;
        int V_Backboundary_u11;
        float DataToDetect = SpatialFFTVelSel_VeloNum_RangeNum_device[tix + tiy * ChirpNum_u11];
        if (tix < (ChirpNum_u11 - 1))
        {
            V_Upboundary_u11 = tix + 1;
        }
        else
        {
            V_Upboundary_u11 = 0;
        }
        if (tix > 0)
        {
            V_Backboundary_u11 = tix - 1;
        }
        else
        {
            V_Backboundary_u11 = ChirpNum_u11 - 1;
        }
        if (tiy == 0)
        {
            isPeak[tix + tiy * ChirpNum_u11] = (DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[tix + (tiy + 1) * ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + (tiy + 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + (tiy + 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + tiy*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + tiy*ChirpNum_u11])  
        }
        else if(tiy == RangeCellNum_u10 - 1)
        {
            isPeak[tix + tiy * ChirpNum_u11] = (DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[tix + (tiy - 1) * ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + (tiy - 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + (tiy - 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + tiy*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + tiy*ChirpNum_u11]);
        }
        else
        {
            isPeak[tix + tiy * ChirpNum_u11] = (DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[tix + (tiy + 1) * ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + (tiy + 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + (tiy + 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[tix + (tiy - 1) * ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + (tiy - 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + (tiy - 1)*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Upboundary_u11 + tiy*ChirpNum_u11] && DataToDetect > SpatialFFTVelSel_VeloNum_RangeNum_device[V_Backboundary_u11 + tiy*ChirpNum_u11]);
        }
    } 
}

__global__ void CFARChM_OS_1D_kernel(uint8_t* IsTarget_1D_R_u1, float* RSNR_u11, uint16_t RangeCellNum_u10, uint16_t ChirpNum_u11, uint8_t ProCellNum_R_u2, uint8_t RefCellNum_1D_u5, uint8_t Loc_OSCFAR_u5, uint8_t* isPeak, float* SpatialFFTVelSel_VelNum_RangeNum_device)
{
    int tix = blockDim.x * blockIdx.x + threadIdx.x;
    int tiy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tix < ChirpNum_u11 && tiy < RangeCellNum_u10)
    {
        if (isPeak[tix + tiy * ChirpNum_u11])
        {
            uint16_t LeftBoundary_u9 = ProCellNum_R_u2 + RefCellNum_1D_u5 - 1;
            uint16_t RightBoundary_u10 = RangeCellNum_u10 - ProCellNum_R_u2 - RefCellNum_1D_u5;
            uint8_t Logic1_u1 = 0;
            uint8_t Logic2_u1 = 0;
            uint8_t RefCellNum_u6 = 0;
            float DataSet_RefCell_u30[32] = {0.0};
            if (LeftBoundary_u9 >= RightBoundary_u10)
            {
                ProCellNum_R_u2 = 1;
                RefCellNum_1D_u5 = 1;
                LeftBoundary_u9 = ProCellNum_R_u2 + RefCellNum_1D_u5 - 1;
                RightBoundary_u10 = RangeCellNum_u10 - ProCellNum_R_u2 - RefCellNum_1D_u5;
            }

            if (tiy > LeftBoundary_u9)
            {
                Logic1_u1 = 1;
            }
            if (tiy < RightBoundary_u10)
            {
                Logic2_u1 = 1;
            }
            
            if (Logic1_u1 == 1 && Logic2_u1 == 1)
            {
                RefCellNum_u6 = 2 * RefCellNum_1D_u5;
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if ((i%2) == 1)
                    {
                        uint16_t leftRefIndex = tiy - ProCellNum_R_u2 - RefCellNum_1D_u5 + i / 2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + leftRefIndex * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t rightRefIndex = tiy + ProCellNum_R_u2 + 1 + i / 2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + rightRefIndex * ChirpNum_u11];
                    }
                }
            }
            else if(Logic2_u1 == 1)
            {
                RefCellNum_u6 = RefCellNum_1D_u5 + tiy - ProCellNum_R_u2 - 1;
                if (RefCellNum_u6 < RefCellNum_1D_u5)
                {
                    RefCellNum_u6 = RefCellNum_1D_u5
                }
                for (uint8_t i = 0; i < RefCellNum_1D_u5; i++)
                {
                    uint16_t rightRefIndex = tiy + ProCellNum_R_u2 + 1 + i;
                    DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + rightRefIndex * ChirpNum_u11];
                }
                for (uint8_t i = RefCellNum_1D_u5; i < RefCellNum_u6; i++)
                {
                    uint16_t leftRefIndex = 1 + i - RefCellNum_1D_u5;
                    DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + leftRefIndex * ChirpNum_u11];
                }
            }
            else
            {
                RefCellNum_u6 = RefCellNum_1D_u5 + RangeCellNum_u10 - tiy - ProCellNum_R_u2 - 1;
                for (uint16_t i = 0; i < RefCellNum_1D_u5; i++)
                {
                    uint16_t leftRefIndex = tiy - ProCellNum_R_u2 - RefCellNum_1D_u5 + i;
                    DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + leftRefIndex * ChirpNum_u11];
                }
                for (uint16_t i = RefCellNum_1D_u5; i < RefCellNum_u6; i++)
                {
                    uint16_t rightRefIndex = tiy + ProCellNum_R_u2 + 1 + i - RefCellNum_1D_u5;
                    SpatialFFTVelSel_VelNum_RangeNum_device[tix + rightRefIndex * ChirpNum_u11];
                }
            }
            if (Loc_OSCFAR_u5 >= RefCellNum_u6)
            {
                Loc_OSCFAR_u5 = floor(double(RefCellNum_u6 / 2) + 1);
            }
            
            float Data_OSLoc_u30;
            Data_OSLoc_u30 = DataSet_RefCell_u30[RefCellNum_u6 - 1];
            uint8_t LeftLoc_u1 = 1;
            uint8_t LocComp_u5 = Loc_OSCFAR_u5 - 1;
            uint8_t middleLoc = floor(double(RefCellNum_u6 / 2));
            if (Loc_OSCFAR_u5 > middleLoc)
            {
                LeftLoc_u1 = 0;
                LocComp_u5 = RefCellNum_u6 - Loc_OSCFAR_u5;
            }
            for (uint8_t i = 0; i < RefCellNum_u6 - 1; i++)
            {
                float Cell_ToCompare_u30 = DataSet_RefCell_u30[i];
                uint8_t IsData_u1 = 0;
                uint8_t Count_u5 = 0;
                uint8_t Data_Equal_u5 = 0;
                uint8_t flag_Not_u1 = 0;
                for (uint8_t j = 0; j < RefCellNum_u6; j++)
                {
                    float Cell_Temp_u30 = DataSet_RefCell_u30[j];
                    if (LeftLoc_u1 == 1)
                    {
                        if (Cell_ToCompare_u30 > Cell_Temp_u30)
                        {
                            Count_u5 = Count_u5 + 1;
                        }
                        
                    }
                    else
                    {
                        if(Cell_ToCompare_u30 < Cell_Temp_u30)
                        {
                            Count_u5 = Count_u5 + 1;
                        }
                    }
                    if (Cell_ToCompare_u30 == Cell_Temp_u30)
                    {
                        Data_Equal_u5 = Data_Equal_u5 + 1;
                    }
                    if(Count_u5 > LocComp_u5)
                    {
                        flag_Not_u1 = 1;
                        break;
                    }
                }
                if (flag_Not_u1 == 0)
                {
                    if (Data_Equal_u5 == 1)
                    {
                        if (Count_u5 == LocComp_u5)
                        {
                            IsData_u1 = 1;
                        }
                        
                    }
                    else
                    {
                        uint8_t NumLess_u5 = Count_u5 + Data_Equal_u5;
                        if (NumLess_u5 > LocComp_u5)
                        {
                            IsData_u1 = 1;
                        }
                    }
                }
                if (IsData_u1 == 1)
                {
                    Data_OSLoc_u30 = Cell_ToCompare_u30;
                    break;
                }
            }
            uint8_t R_Threshold_Num_u6 = 32;
            uint8_t thIdx = floor(double(tiy / (RangeCellNum_u10/R_Threshold_Num_u6)))                    
            float Threshold2_u39 = 5 * Data_OSLoc_u30;
            float CellToDetect_u32 = SpatialFFTVelSel_VelNum_RangeNum_device[tix + tiy * ChirpNum_u11] * 2 * 2;
            if (CellToDetect_u32 > Threshold2_u39)
            {
                IsTarget_1D_R_u1[tix + tiy * ChirpNum_u11] = 1;
                RSNR_u11[tix + tiy * ChirpNum_u11] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + tiy * ChirpNum_u11] / Data_OSLoc_u30;
            } 
        }
    }
}

__global__ void CFARChm_OS_1D_V_kernel(uint8_t* IsTarget_1D_V_u1, float* VSNR_u11, uint8_t LogicTestFlag_u1, uint16_t RangeCellNum_u10, uint16_t ChirpNum_u11, uint8_t ProCellNum_V_u2, uint8_t RefCellNum_1D_u5, uint8_t Loc_OSCFAR_u5, uint8_t* IsTarget_1D_R_u1, float* SpatialFFTVelSel_VelNum_RangeNum_device)
{
    int tix = blockDim.x * blockIdx.x + threadIdx.x;
    int tiy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tix < ChirpNum_u11 && tiy < RangeCellNum_u10)
    {
        if (IsTarget_1D_R_u1[tix + tiy * ChirpNum_u11] || LogicTestFlag_u1 == 1)
        {
            uint8_t Logic1_u1 = 0;
            uint8_t Logic2_u1 = 0;
            uint8_t Logic3_u1 = 0;
            uint8_t Logic4_u1 = 0;
            uint8_t Logic5_u1 = 0;
            uint16_t UpBoundary_u9 = 1 + ProCellNum_V_u2 + RefCellNum_1D_u5;
            uint16_t DownBoundary_u10 = ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5;
            uint16_t RefCellNum_u6 = 2 * RefCellNum_1D_u5;
            float DataSet_RefCell_u30[32] = {0.0};
            if(UpBoundary_u9 >= DownBoundary_u10)
            {
                ProCellNum_V_u2 = 1;
                RefCellNum_1D_u5 = 1;
                UpBoundary_u9 = 1 + ProCellNum_V_u2 + RefCellNum_1D_u5;
                DownBoundary_u10 = ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5;
            }
            if (tix >= 0 && tix <= 0 + ProCellNum_V_u2)
            {
                Logic1_u1 = 1;
            }
            if(tix > (0 + ProCellNum_V_u2) && tix < (0 + ProCellNum_V_u2 + RefCellNum_1D_u5))
            {
                Logic2_u1 = 1;
            }
            if (tix >= 0 + ProCellNum_V_u2 + RefCellNum_1D_u5 && tix <= ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5 - 1)
            {
                Logic3_u1 = 1;
            }
            if (tix > ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5 - 1 && tix < ChirpNum_u11 - ProCellNum_V_u2 - 1)
            {
                Logic4_u1 = 1;
            }
            if (tix >= ChirpNum_u11 - ProCellNum_V_u2 - 1)
            {
                Logic5_u1 = 1;
            }
            if (Logic1_u1 == 1)
            {
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if (i % 2 == 1)
                    {
                        uint16_t upIndex = ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5 + tix + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[upIndex + tiy * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t downIndex = tix + ProCellNum_V_u2 + 1 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[downIndex + tiy * ChirpNum_u11];
                    }
                }
                
            }
            else if(Logic2_u1 == 1)            
            {
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if (i % 2 == 1)
                    {
                        uint16_t upIndex = (ChirpNum_u11 - ProCellNum_V_u2 - RefCellNum_1D_u5 + tix + i/2) % ChirpNum_u11;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[upIndex + tiy * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t downIndex = tix + ProCellNum_V_u2 + 1 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[downIndex + tiy * ChirpNum_u11];
                    }
                }
                
            }
            else if(Logic3_u1 == 1)
            {
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if (i % 2 == 1)
                    {
                        uint16_t upIndex = tix - ProCellNum_V_u2 - RefCellNum_1D_u5 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[upIndex + tiy * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t downIndex = tix + ProCellNum_V_u2 + 1 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[downIndex + tiy * ChirpNum_u11];
                    }
                }
            }
            else if(Logic4_u1 == 1)
            {
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if (i % 2 == 1)
                    {
                        uint16_t upIndex = tix - ProCellNum_V_u2 - RefCellNum_1D_u5 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[upIndex + tiy * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t downIndex = (tix + ProCellNum_V_u2 + 1 + i/2) % ChirpNum_u11;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[downIndex + tiy * ChirpNum_u11];
                    }
                }
            }
            else if(Logic5_u1 == 1)
            {
                for (uint8_t i = 0; i < RefCellNum_u6; i++)
                {
                    if (i % 2 == 1)
                    {
                        uint16_t upIndex = tix - ProCellNum_V_u2 - RefCellNum_1D_u5 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[upIndex + tiy * ChirpNum_u11];
                    }
                    else
                    {
                        uint16_t downIndex = tix + ProCellNum_V_u2 + 1 - ChirpNum_u11 + i/2;
                        DataSet_RefCell_u30[i] = SpatialFFTVelSel_VelNum_RangeNum_device[downIndex + tiy * ChirpNum_u11];
                    }
                }
            }
            if (Loc_OSCFAR_u5 >= RefCellNum_u6)
            {
                Loc_OSCFAR_u5 = floor(double(RefCellNum_u6 / 2) + 1);
            }
            
            float Data_OSLoc_u30;
            Data_OSLoc_u30 = DataSet_RefCell_u30[RefCellNum_u6 - 1];
            uint8_t LeftLoc_u1 = 1;
            uint8_t LocComp_u5 = Loc_OSCFAR_u5 - 1;
            uint8_t middleLoc = floor(double(RefCellNum_u6 / 2));
            if (Loc_OSCFAR_u5 > middleLoc)
            {
                LeftLoc_u1 = 0;
                LocComp_u5 = RefCellNum_u6 - Loc_OSCFAR_u5;
            }
            for (uint8_t i = 0; i < RefCellNum_u6 - 1; i++)
            {
                float Cell_ToCompare_u30 = DataSet_RefCell_u30[i];
                uint8_t IsData_u1 = 0;
                uint8_t Count_u5 = 0;
                uint8_t Data_Equal_u5 = 0;
                uint8_t flag_Not_u1 = 0;
                for (uint8_t j = 0; j < RefCellNum_u6; j++)
                {
                    float Cell_Temp_u30 = DataSet_RefCell_u30[j];
                    if (LeftLoc_u1 == 1)
                    {
                        if (Cell_ToCompare_u30 > Cell_Temp_u30)
                        {
                            Count_u5 = Count_u5 + 1;
                        }
                        
                    }
                    else
                    {
                        if(Cell_ToCompare_u30 < Cell_Temp_u30)
                        {
                            Count_u5 = Count_u5 + 1;
                        }
                    }
                    if (Cell_ToCompare_u30 == Cell_Temp_u30)
                    {
                        Data_Equal_u5 = Data_Equal_u5 + 1;
                    }
                    if(Count_u5 > LocComp_u5)
                    {
                        flag_Not_u1 = 1;
                        break;
                    }
                }
                if (flag_Not_u1 == 0)
                {
                    if (Data_Equal_u5 == 1)
                    {
                        if (Count_u5 == LocComp_u5)
                        {
                            IsData_u1 = 1;
                        }
                        
                    }
                    else
                    {
                        uint8_t NumLess_u5 = Count_u5 + Data_Equal_u5;
                        if (NumLess_u5 > LocComp_u5)
                        {
                            IsData_u1 = 1;
                        }
                    }
                }
                if (IsData_u1 == 1)
                {
                    Data_OSLoc_u30 = Cell_ToCompare_u30;
                    break;
                }
            }
            uint8_t thIdx = floor(double(tix / (ChirpNum_u11/16)))                    
            float Threshold2_u39 = 8 * Data_OSLoc_u30;
            float CellToDetect_u32 = SpatialFFTVelSel_VelNum_RangeNum_device[tix + tiy * ChirpNum_u11] * 2 * 2;
            if (CellToDetect_u32 > Threshold2_u39)
            {
                IsTarget_1D_R_u1[tix + tiy * ChirpNum_u11] = 1;
                VSNR_u11[tix + tiy * ChirpNum_u11] = SpatialFFTVelSel_VelNum_RangeNum_device[tix + tiy * ChirpNum_u11] / Data_OSLoc_u30;
            } 
        }
    }
}

__global__ postProcess_kernel(uint16_t ChirpNum_u11, uint16_t RangeCellNum_u10, uint16_t Index_Chirp_NotMove_OSCFAR_u11, uint16_t Threshold_RangeDim_For_2D_OSCFAR_u9, uint8_t* IsTarget_1D_R_u1, uint8_t* IsTarget_1D_V_u1, float* SpatialFFTVelSel_VeloNum_RangeNum_device)
{
    int tix = blockDim.x * blockIdx.x + threadIdx.x;
    int tiy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tix < ChirpNum_u11 && tiy < RangeCellNum_u10)
    {
        /* code */
    }
}

void_winR_process(float* WinRout_RangeSampleNum_ChirpNum_RxNum_device, int16_t* radarInputdata_device, float* win_Coef_device, uint8_t& fft_win_type, uint16_t& RangeSampleNum, uint16_t& ChirpNum, uint16_t& RxNum)
{
    dim3 block_win(256, 1, 1);
    dim3 grid_win((RangeSampleNum + block_win.x - 1) / block_win.x,1,1);
    dim3 block_fftr(16,1,32);
    dim3 grid_fftr((RangeSampleNum + block_fftr.x - 1) / block_fftr.x, (RxNum + block_fftr.y - 1)/block_fftr.y, (ChirpNum + block_fftr.z - 1)/block_fftr.z);

    switch (fft_win_type)
    {
    case 0:
        rectWin_kernel<<<grid_win, block_win>>>(win_Coef_device, RangeSampleNum);
        break;
    case 1:
        hanningWin_kernel<<<grid_win, block_win>>>(win_Coef_device, RangeSampleNum);
        break;
    case 2:
        hammingWin_kernel<<<grid_win, block_win>>>(win_Coef_device, RangeSampleNum);
        break;
    default:
        break;
    }

    addWin_kernel<<<block_fftr, grid_fftr>>>(WinRout_RangeSampleNum_ChirpNum_RxNum_device,radarInputdata_device,win_Coef_device,RangeSampleNum,ChirpNum,RxNum);
}

void func_fftR_process(float2* CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device, float* WinRout_RangeSampleNum_ChirpNum_RxNum_device, uint16_t& RangeSampleNum, uint16_t& VelocityNum, uint16_t& RxNum)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, RangeSampleNum, HIPFFT_R2C, VelocityNum*RxNum)
    hipfftExecR2C(plan, WinRout_RangeSampleNum_ChirpNum_RxNum_device, CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device);
    hipfftDestroy(plan);
}

void func_dataReshape_afterFFTR(float2* CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device, float2* CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device, uint16_t* all_tx_seq_pos, uint16_t& TxNum, uint16_t& RxNum, uint16_t& TxReuseNum, uint16_t& CoarseRangeNum)
{
    uint16_t velocityNum = TxNum * TxReuseNum;
    dim3 block(16,64,1);
    dim3 grid((CoarseRangeNum + block.x - 1) / block.x, (velocityNum + block.y - 1) / block.y, RxNum);
    uint16_t* all_tx_seq_pos_device;
    hipMalloc(all_tx_seq_pos_device, velocityNum * sizeof(uint16_t));
    hipMemcpy(all_tx_seq_pos_device, all_tx_seq_pos, velocityNum * sizeof(uint16_t), hipMemcpyHostToDevice);
    dataReshape_afterFFTR_kernel<<<grid, block>>>(CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device,CoarseRangeFFT_ValidCoarseRangeBinNum_ChirpNum_RxNum_device,all_tx_seq_pos_device,TxNum,RxNum,TxReuseNum,CoarseRangeNum);
    hipFree(all_tx_seq_pos_device);
}

void func_winD_process(float2* WinDout_ChirpNum_RangeSampleNum_MIMONum_device, float2* CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device, float* win_Coef_device, uint8_t& fft_win_type, uint16_t& VelocityNum, uint16_t& CoarseRangeNum, uint16_t& MimoNum)
{
    dim3 block_win(256,1,1);
    dim3 grid_win((VelocityNum + block_win.x - 1) / block_win.x,1,1);
    dim3 block_fftd(64,16,1);
    dim3 grid_fftd((VelocityNum + blockfftd.x - 1) / block_fftd.x, (CoarseRangeNum + blockfftd.y - 1)/blockfftd.y, MimoNum);

    switch (fft_win_type)
    {
    case 0:
        rectWin_kernel<<<grid_win,block_win>>>(win_Coef_device,VelocityNum);
        break;
    case 1:
        hanningWin_kernel<<<grid_win,block_win>>>(win_Coef_device,VelocityNum);
        break;
    case 2:
        hammingWin_kernel<<<grid_win,block_win>>>(win_Coef_device,VelocityNum);
        break;
    default:
        break;
    }

    addWinD_kernel<<<grid_fftd, block_fftd>>>(WinDout_ChirpNum_RangeSampleNum_MIMONum_device,CoarseRangeFFT_ChirpNum_RangeSampleNum_MIMONum_device,win_Coef_device,VelocityNum,CoarseRangeNum,MimoNum);
}

void func_fftD_process(float2* FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device, float2* WinDout_ChirpNum_RangeSampleNum_MIMONum_device, uint16_t& VelocityNum, uint16_t& CoarseRangeNum, uint16_t& MIMONum)
{
    hipfftHandle plan;
    int Batch = CoarseRangeNum * MIMONum;
    hipfftPlan1d(&plan,VelocityNum,HIPFFT_C2C,Batch);
    hipfftExecC2C(plan, WinDout_ChirpNum_RangeSampleNum_MIMONum_device,FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device,HIPFFT_FORWARD);
    hipfftDestroy(plan);

    dim3 block(64, 16, 1);
    dim3 grid((VelocityNum/2 + block.x - 1) / block.x, (CoarseRangeNum + block.y - 1)/block.y, MIMONum);
    fftshift<<<grid,block>>>(FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device, VelocityNum, CoarseRangeNum, MIMONum);
}

void func_Spatial_Reorder(float2* result_xNum_yNum_VeloFFTNum_RangeNum, float2* FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device, uint8_t& Array_option, uint16_t& MimoNum, uint16_t* pos_in_mat, uint16_t& VirtArrHorGridLen, uint16_t& VirtArrVertGridLen, uint16_t& VelocityNum,uint16_t& CoarseRangeNum)
{
    uint16_t* pos_in_mat_device;
    hipMalloc(&pos_in_mat_device, MimoNum * 2 * sizeof(uint16_t));
    hipMemcpy(pos_in_mat_device, pos_in_mat, MimoNum * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
    switch (Array_option)
    {
    case 0:
        ;
        break;
    case 1:
        ;
        break;
    case 2:
        ;
        break;
    case 3:
        dim3 block(64,16,1);
        dim3 grid((VelocityNum/2 + block.x - 1) / block.x, (CoarseRangeNum + block.y - 1)/block.y, MimoNum);
        spatial_reorder_kernel<<<grid,block>>>(result_xNum_yNum_VeloFFTNum_RangeNum,FFT2D_VeloFFTNum_CoarseRangeBinNum_MIMONum_device,pos_in_mat_device,MimoNum,VelocityNum,CoarseRangeNum,VirtArrHorGridLen,VirtArrVertGridLen);
        break;
    default:
        break;
    }

    hipFree(pos_in_mat_device);
}


void func_winA_process(float2* winAout_xNum_yNum_VeloFFTNum_RangeNum_device, float2* result_xNum_yNum_VeloFFTNum_RangeNum_device, float* win_Coef_device, uint8_t& fft_win_type, uint16_t& VirtArrHorGridLen, uint16_t VirtArrVertGridLen, uint16_t& VelocityNum, uint16_t& CoarseRangeNum, uint16_t& AngleHorNum)
{
    dim3 block_win(27,1,1);
    dim3 grid_win((VirtArrHorGridLen + block_win.x - 1)/ block_win.x, 1,1);
    dim3 block_ffta(1,64,16);
    dim3 grid_ffta((VirtArrHorGridLen + block_ffta.x - 1)/block_ffta.x, (VelocityNum + block_ffta.y - 1)/block_ffta.y,(CoarseRangeNum+block_ffta.z-1)/block_ffta.z);
    switch (fft_win_type)
    {
    case 0:
        rectWin_kernel<<<grid_win,block_win>>>(win_Coef_device, VirtArrHorGridLen);
        break;
    case 1:
        hanningWin_kernel<<<grid_win,block_win>>>(win_Coef_device, VirtArrHorGridLen);
        break;
    case 2:
        hammingWin_kernel<<<grid_win,block_win>>>(win_Coef_device, VirtArrHorGridLen);
        break;
    default:
        break;
    }

    addWinA_kernel<<<grid_ffta,block_ffta>>>(winAout_xNum_yNum_VeloFFTNum_RangeNum_device,result_xNum_yNum_VeloFFTNum_RangeNum_device,win_Coef_device,VirtArrHorGridLen,VirtArrVertGridLen,VelocityNum,CoarseRangeNum,AngleHorNum);
}

void func_fftA_process(float2* SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, float2* winAout_xNum_yNum_VeloFFTNum_RangeNum_device, uint16_t& AngleHorNum, uint16_t& VirtArrVertGridLen, uint16_t& VelocityNum, uint16_t& CoarseRangeNum)
{
    hipfftHandle plan;
    int Batch = VirtArrVertGridLen * VelocityNum * CoarseRangeNum;
    hipfftPlan1d(&plan, AngleHorNum, HIPFFT_C2C, Batch);
    hipfftExecC2C(plan, winAout_xNum_yNum_VeloFFTNum_RangeNum_device, SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, HIPFFT_FORWARD);
    hipfftDestroy(plan);

    dim3 block(1, 64, 16);
    dim3 grid((AngleHorNum + block.x - 1)/block.x, (VelocityNum + block.y - 1)/block.y, (CoarseRangeNum + block.z - 1)/block.z);
    fftshift_ffta<<<grid,block>>>(SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device,AngleHorNum,VirtArrVertGridLen,VelocityNum,CoarseRangeNum);
}

void func_abs_process(float* SpatialFFTA_ABS_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, float2* SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, uint16_t& AngleHorNum, uint16_t& VirtArrVertGridLen, uint16_t& VelocityNum, uint16_t CoarseRangeNum)
{
    dim3 block(1,64,16);
    dim3 grid((AngleHorNum + block.x - 1)/block.x, (VelocityNum + block.y - 1)/block.y, (CoarseRangeNum + block.z - 1)/block.z);
    ffta_abs_kernel<<<grid,block>>>(SpatialFFTA_ABS_AngleHorNum_yNum_VeloFFTNum_RangeNum_device,SpatialFFTA_AngleHorNum_yNum_VeloFFTNum_RangeNum_device,AngleHorNum,VirtArrVertGridLen,VelocityNum,CoarseRangeNum);
}

void func_cfar3d_cal_across_ArbitaryDim(float* SpatialFFTVelSel_VeloNum_RangeNum_device, float* SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device, uint8_t& SqueezeDim, uint8_t& cfar_include_order, uint8_t& cfar_exclude_order, float& snr_dB_different_dim, uint8_t& Switch3DMode, uint16_t& AngleHorNum, uint16_t& VelocityNum, uint16_t& CoarseRangeNum)
{
    dim3 block(128,4,2);
    dim3 grid((AngleHorNum + block.x - 1)/block.x, (VelocityNum + block.y - 1)/block.y, (CoarseRangeNum + block.z - 1)/block.z);
    cfar3d_cal_across_ArbitaryDim_kernel<<<grid,block>>>(SpatialFFTVelSel_VeloNum_RangeNum_device,SpatialFFTA_ABS_Mean_AngleHorNum_yNum_VeloFFTNum_RangeNum_device,AngleHorNum,VelocityNum,CoarseRangeNum);
}

void func_PeakSearch_And_CFAR_2D_Cross(uint16_t& TarNum_Detected, uint16_t* peak_R, uint16_t* peak_V, float* peak_Val, float* peak_SNR, DetPara& det_para, float* SpatialFFTVelSel_VeloNum_RangeNum_device)
{
    uint8_t *isPeak;
    hipMalloc(&isPeak, det_para.ChirpNum_u11 * det_para.RangeCellNum_u10 * sizeof(uint8_t));
    dim3 block(64,16);
    dim3 grid((det_para.ChirpNum_u11 + block.x - 1)/block.x, (det_para.RangeCellNum_u10 + block.y - 1)/block.y);
    peak_search_kernel<<<grid,block>>>(isPeak,SpatialFFTVelSel_VeloNum_RangeNum_device,det_para.DetectCell_RIndex_Min_u10,det_para.DetectCell_RIndex_Max_u10, det_para.DetectCell_VIndex_Min_u11, det_para.DetectCell_VIndex_Max_u11,det_para.ChirpNum_u11,det_para.RangeCellNum_u10);
    hipDeviceSynchronize();

    dim3 blockd(16,4);
    dim3 gridd((det_para.ChirpNum_u11 + blockd.x - 1)/blockd.x, (det_para.RangeCellNum_u10 + blockd.y - 1)/blockd.y);
    uint8_t* IsTarget_1D_R_u1;
    float* RSNR_u11;
    hipMalloc(&IsTarget_1D_R_u1, det_para.ChirpNum_u11 * det_para.RangeCellNum_u10 * sizeof(uint8_t));
    hipMalloc(&RSNR_u11, det_para.ChirpNum_u11 * det_para.RangeCellNum_u10 * sizeof(float));
    CFARChM_OS_1D_kernel<<<gridd,blockd>>>(IsTarget_1D_R_u1,RSNR_u11,det_para.RangeCellNum_u10,det_para.ChirpNum_u11,det_para.cfar_para.ProCellNum_R_u2,det_para.cfar_para.RefCellNum_1D_u5,det_para.Lod_OSCFAR_u5,isPeak,SpatialFFTVelSel_VeloNum_RangeNum_device)
    hipDeviceSynchronize();

    uint8_t* IsTarget_1D_V_u1;
    float* VSNR_u11;
    hipMalloc(&IsTarget_1D_V_u1, det_para.ChirpNum_u11 * det_para.RangeCellNum_u10 * sizeof(uint8_t));
    hipMalloc(&VSNR_u11, det_para.ChirpNum_u11 * det_para.RangeCellNum_u10 * sizeof(float));
    CFARChM_OS_1D_V_kernel<<<gridd,blockd>>>(IsTarget_1D_V_u1,VSNR_u11,det_para.LogicTestFlag_u1,det_para.RangeCellNum,det_para.ChirpNum_u11,det_para.cfar_para.ProCellNum_V_u2,det_para.cfar_para.RefCellNum_1D_u5,det_para.Loc_OSCFAR_u5,IsTarget_1D_R_u1, SpatialFFTVelSel_VeloNum_RangeNum_device);

    postProcess_kernel<<<grid,block>>>(det_para.ChirpNum_u11,det_para.RangeCellNum_u10,det_para.Index_Chirp_NotMove_OSCFAR_u11, det_para.Threshold_RangeDim_For_2D_OSCFAR_u9,IsTarget_1D_R_u1,IsTarget_1D_V_u1,SpatialFFTVelSel_VeloNum_RangeNum_device);
    hipDeviceSynchronize();

    // 将结果memcpy到主机端，赋给输出参数

    // free掉内存
}